#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include "CostVolume.cuh"

// 2D float texture
static texture<float, hipTextureType2D, hipReadModeElementType> current_imageTexRef;

static __global__ void updateCostVolume(float* K, float* Kinv, float* Tmr,
										int rows, int cols,
										float near, float far, int layers, int layerStep,
										float* Cost, float count,
										float* Cmin, float* Cmax, float* CminIdx,
										float* reference_image, float* current_image)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int i = x + y*cols;

	const float ur = x;
	const float vr = y;

	const float depthStep = (near - far)/(layers-1);

	float Ir = reference_image[i];

	int	  minl = layers-1; // TODO set to layers?
	float Cost_min = 1e+30, Cost_max = 0.0;
	for(int l=layers-1; l >= 0; l--) { // TODO march from front to back, i.e., l = layers -> 0 and check results. 
		float d = far + float(l)*depthStep;
		// 0 1 2
		// 3 4 5
		// 6 7 8
		float zr = 1.0/d; // divide by 0 is evaluated as Inf, as per IEEE-754
		float xr = (Kinv[0]*ur + Kinv[2])*zr;
		float yr = (Kinv[4]*vr + Kinv[5])*zr;
		//  0  1  2  3
		//  4  5  6  7
		//  8  9 10 11
		// 12 13 14 15
		float xm = Tmr[0]*xr + Tmr[1]*yr + Tmr[2]*zr  + Tmr[3];
		float ym = Tmr[4]*xr + Tmr[5]*yr + Tmr[6]*zr  + Tmr[7];
		float zm = Tmr[8]*xr + Tmr[9]*yr + Tmr[10]*zr + Tmr[11];
		// 0 1 2
		// 3 4 5
		// 6 7 8
		float um = K[0]*(xm/zm) + K[2];
		float vm = K[4]*(ym/zm) + K[5];

		if( (um > float(cols)) || (um < 0.0f) || (vm > float(rows)) || (vm < 0.0f) )
			continue;

		float Im = tex2D(current_imageTexRef, um, vm);

		float rho = fabsf(Ir - Im);
		Cost[i+l*layerStep] = (Cost[i+l*layerStep]*(count-1) + rho) / count; // TODO: maintain per pixel count? Not necessary. 
		float Cost_l = Cost[i+l*layerStep];
		if(Cost_l <= Cost_min) {
			Cost_min = Cost_l;
			minl = l;
		}
		Cost_max = fmaxf(Cost_l, Cost_max);
	}

	Cmin[i]	   = Cost_min;
	CminIdx[i] = far + float(minl)*depthStep; // scaling is done when used in DepthEstimator::optimize
	Cmax[i]	   = Cost_max;

	// sublayer sampling as the minimum of the parabola with the 2 points around (minz, minv)
	if(minl == 0 || minl == layers-1) // first or last was best
		return;

	float A = far + float(minl-1)*depthStep;
	float B = CminIdx[i];
	float C = far + float(minl+1)*depthStep;
	float delta = ((A+C)==2*B)? 0.0f : ((A-C)*depthStep)/(2*(A-2*B+C));
	delta = (fabsf(delta) > depthStep)? 0.0f : delta;
	CminIdx[i] += delta;
}

void updateCostVolumeCaller(float* K, float* Kinv, float* Tmr,
							int rows, int cols, int imageStep,
							float near, float far, int layers, int layerStep,
							float* Cdata, float count,
							float* Cmin, float* Cmax, float* CminIdx,
							float* reference_image, float* current_image)
{
	dim3 dimBlock(16, 16);
	dim3 dimGrid((cols + dimBlock.x - 1) / dimBlock.x,
				 (rows + dimBlock.y - 1) / dimBlock.y);

	// Set texture reference parameters
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	current_imageTexRef.normalized	  = false;
	current_imageTexRef.addressMode[0] = hipAddressModeClamp; // out of border references return first or last element
	current_imageTexRef.addressMode[1] = hipAddressModeClamp;
	current_imageTexRef.filterMode	  = hipFilterModeLinear;

	// Bind current_image to the texture reference
	size_t offset;
	hipBindTexture2D(&offset, current_imageTexRef, current_image, channelDesc, cols, rows, imageStep);

	hipDeviceSynchronize();
	cudaSafeCall(hipGetLastError());

	updateCostVolume<<<dimGrid, dimBlock>>>(K, Kinv, Tmr,
											rows, cols,
											near, far, layers, layerStep,
											Cdata, count,
											Cmin, Cmax, CminIdx,
											reference_image, current_image);
	hipDeviceSynchronize();
	cudaSafeCall(hipGetLastError());
	hipUnbindTexture(current_imageTexRef);
}
