#include "hip/hip_runtime.h"
#include <assert.h>
#include <iostream>
#include <opencv2/core/core.hpp>
#include "CostVolume.cuh"

// Using a different version of the accelerated search method:
// a_min must lie between [(d_i-d_min), (d_i+d_min)]

#define SET_start_layer()                                   \
    float d_start = di - r;                                 \
    layer = lrintf(floorf((d_start - far)/depthStep)) - 1;  \
    layer = (layer<0)? 0 : layer;                           \
    start_layer = layer;                                    \

#define SET_end_layer()                                     \
    float d_end = di + r;                                   \
    layer = lrintf(ceilf((d_end - far)/depthStep)) + 1;     \
    layer = (layer>(layers-1))? (layers-1) : layer;         \
    end_layer = layer;                                      \

__device__
static inline float Eaux(float theta, float di, float aIdx, float far, float depthStep, float lambda, float costval)
{
	float ai = far + float(aIdx)*depthStep;
	return (0.5f/theta)*((di-ai)*(di-ai)) + lambda*costval; // TODO beware float substraction
}

static __global__ void minimizeA(float* cost, int rows, int cols,
								 float* a, float* d,
								 float*d_Cmin, float*C_min, float*C_max,
								 float far, float near, int layers,
								 float theta, float lambda)
{
	// thread coordinate
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int i = x + y*cols;

	const float depthStep = (near - far) / (layers - 1);
	const int	layerStep = rows*cols;
	const float di		  = d[i];

	float r = sqrtf(2*theta*lambda*(C_max[i] - C_min[i]));

	int minl = 0;
	float Eaux_min = 1e+30;
	int layer, start_layer, end_layer;
	SET_start_layer();
	SET_end_layer();
	for(int l=start_layer; l<=end_layer; l++) {
		float c = Eaux(theta, di, l, far, depthStep, lambda, cost[i+l*layerStep]);
		if(c < Eaux_min) {
			Eaux_min = c;
			minl = l;
		}
	}

	a[i] = far + float(minl)*depthStep;

	if(minl == 0 || minl == layers-1) // first or last was best
		return;

	// sublayer sampling as the minimum of the parabola with the 2 points around (minl, Eaux_min)
	float A = Eaux(theta, di, minl-1, far, depthStep, lambda, cost[i+(minl-1)*layerStep]);
	float B = Eaux_min;
	float C = Eaux(theta, di, minl+1, far, depthStep, lambda, cost[i+(minl+1)*layerStep]);
	float delta = ((A+C)==2*B)? 0.0f : ((A-C)*depthStep)/(2*(A-2*B+C));
	delta = (fabsf(delta) > depthStep)? 0.0f : delta;
	a[i] += delta;
}

void minimizeACaller(float *cdata, int rows, int cols,
					 float *a, float *d,
					 float*d_Cmin, float*C_min, float*C_max,
					 float far, float near, int layers,
					 float theta, float lambda)
{
	dim3 dimBlock(16, 16);
	dim3 dimGrid((cols + dimBlock.x - 1) / dimBlock.x,
				 (rows + dimBlock.y - 1) / dimBlock.y);

	minimizeA<<<dimGrid, dimBlock>>>(cdata, rows, cols,
									 a, d,
									 d_Cmin, C_min, C_max,
									 far, near, layers,
									 theta, lambda);

	hipDeviceSynchronize();
	cudaSafeCall( hipGetLastError() );
}
